/*****************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations        *
 *  Copyright (C) 2000 - 2018  CP2K developers group                         *
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "acc_cuda_error.h"

/****************************************************************************/
int cuda_error_check (hipError_t hipError_t){
  if (hipError_t != hipSuccess){
      printf ("CUDA Error: %s\n", hipGetErrorString (hipError_t));
      return -1;
    }
  return 0;
};

extern "C" void acc_clear_errors () {
  hipGetLastError();
};

//EOF
